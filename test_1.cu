
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <iostream>

template <typename T, int n>
struct Vec {
  T elems[n];
  __device__ T& operator[](int i) {
    return elems[i];
  }
};
using I4 = Vec<int, 4>;
using FragA = Vec<half2, 4>;
using FragB = Vec<half2, 2>;
using FragC = Vec<float, 4>;
using FragS = Vec<half2, 1>; // quantization scales

int slice_iters = 64;
int stages = 4;
int b_sh_wr_iters = 2;
int thread_m_blocks = 4;

int main() {
  /*Vec<half2, 4>*/ FragA frag_a[2][thread_m_blocks/*4*/];
  /*Vec<int, 4>*/   I4 frag_b_quant[2];
  /*Vec<float, 4>*/ FragC frag_c[thread_m_blocks/*4*/][4][2];
  /*Vec<half2, 1>*/ FragS frag_s[2][4];

  while (slice_iters/* one iter for a tile */) {
    // We unroll over both the global fetch and the register load pipeline to ensure all shared memory accesses are
    // static. Note that both pipelines have even length meaning that the next iteration will always start at index 0.
    #pragma unroll
    for (int pipe = 0; pipe < stages/*4*/;) {
      #pragma unroll
      for (int k = 0; k < b_sh_wr_iters/*2*/; k++) { // call 64 mma inst in total.
        //fetch_to_registers(k + 1, pipe % stages/*4*/);
        // k 的range是 0和1
        if (k == b_sh_wr_iters - 2 /*k=0*/) {
          //fetch_to_shared((pipe + stages/*4*/ - 1) % stages/*4*/, pipe, slice_iters >= stages/*4*/);
          pipe++;
          //wait_for_stage();
        }
        //!!! when k==1, no pipe++
        //matmul(k);
        printf("matmul 32 mma. pipe=%d k=%d\n", pipe, k);
      }
      slice_iters--;
      if (slice_iters == 0)
        break;
    }
    printf("----\n");
  }
}
